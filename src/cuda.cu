#include "hip/hip_runtime.h"
// nvcc cuda.cu -o cuda.out -gencode=arch=compute_75,code=compute_75 -O3
#include <hip/hip_runtime.h>
#include <stdlib.h>

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <string>

#include "config.h"
#include "util.h"

#define BLOCK_SIZE 1024

__global__ void wakeGPU(int reps);
__global__ void floydWarshallKernel(int k, int *matrix, int n);

void floydWarshall(int *matrix, int n, int threadsPerBlock);

int main(int argc, char *argv[]) {
   int n, density, threadsPerBlock;

   if (argc <= 3) {
      n = DEFAULT;
      density = 100;
      threadsPerBlock = BLOCK_SIZE;
   } else {
      n = atoi(argv[1]);
      density = atoi(argv[2]);
      threadsPerBlock = atoi(argv[3]);
   }

   int *matrix = (int *)malloc(n * n * sizeof(int));

   populateMatrix(matrix, n, density);

   if (PRINTABLE) {
      printf("*** Adjacency matrix:\n");
      showDistances(matrix, n);
   }

   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);

   for (int i = 0; i < 3; i++) {
      floydWarshall(matrix, n, threadsPerBlock);
   }
   hipEventRecord(start);
   for (int i = 0; i < 10; i++) {
      floydWarshall(matrix, n, threadsPerBlock);
   }
   hipEventRecord(stop);
   hipEventSynchronize(stop);
   float accum;
   hipEventElapsedTime(&accum, start, stop);
   if (PRINTABLE) {
      printf("*** The solution is:\n");
      showDistances(matrix, n);
   }

   // calculate theoretical occupancy
   int maxActiveBlocksPerSM;
   hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocksPerSM,
                                                 floydWarshallKernel, threadsPerBlock,
                                                 0);

   int device;
   hipDeviceProp_t props;
   hipGetDevice(&device);
   hipGetDeviceProperties(&props, device);

   float occupancy = (maxActiveBlocksPerSM * threadsPerBlock / props.warpSize) /
                     (float)(props.maxThreadsPerMultiProcessor /
                             props.warpSize);

   // printf("maxActiveBlocksPerSM: %d, warpSize: %d, maxThreadsPerMultiProcessor: %d\n", maxActiveBlocksPerSM, props.warpSize, props.maxThreadsPerMultiProcessor);
   printf("[GPGPU] DIM: %d, threadsPerBlock: %d, Theoretical occupancy: %lf, Total elapsed time %.2f ms\n", n, threadsPerBlock, occupancy, accum / 10);
   free(matrix);

   return 0;
}

void floydWarshall(int *matrix, const int n, int threadsPerBlock) {
   int *deviceMatrix;
   int size = n * n * sizeof(int);

   hipMalloc((int **)&deviceMatrix, size);
   hipMemcpy(deviceMatrix, matrix, size, hipMemcpyHostToDevice);

   dim3 dimGrid((n + threadsPerBlock - 1) / threadsPerBlock, n);

   hipFuncSetCacheConfig(reinterpret_cast<const void*>(floydWarshallKernel), hipFuncCachePreferL1);
   for (int k = 0; k < n; k++) {
      floydWarshallKernel<<<dimGrid, threadsPerBlock>>>(k, deviceMatrix, n);
   }
   hipDeviceSynchronize();

   hipMemcpy(matrix, deviceMatrix, size, hipMemcpyDeviceToHost);
   hipFree(deviceMatrix);

   hipError_t err = hipGetLastError();
   if (err != hipSuccess) {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
      exit(EXIT_FAILURE);
   }
}

__global__ void floydWarshallKernel(int k, int *matrix, int n) {
   int i = blockDim.y * blockIdx.y;
   int j = blockDim.x * blockIdx.x + threadIdx.x;

   if (j < n) {
      __shared__ int ik;
      if (threadIdx.x == 0) {
         ik = matrix[i * n + k];
      }
      __syncthreads();

      int newPath = ik + matrix[k * n + j];
      int oldPath = matrix[i * n + j];
      if (oldPath > newPath) {
         matrix[i * n + j] = newPath;
      }
   }
}
